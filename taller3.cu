#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <sys/time.h>
#include <omp.h>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <stdio.h>
#include <string>
#include <math.h>
#include <opencv2/core/core.hpp>

using namespace std;
using namespace cv;
using std::ofstream;

//Se encarga de realiza el algoritmo de sobel a la imagen
__global__ void sobel(unsigned char *d_imgGray, unsigned char *d_imgSobel, int cols, int rows, int numberElements, int totalThreads)
{

    int offSet = cols * 3 + 3;
    int YoffSet = cols * 3;
    int x;

    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
if (index==0){
	
__shared__ Kernel[3][3] = {
                 {-1, 0, 1},
                 {-2, 0, 2},
                 {-1, 0, 1}
    };
__shared__ Kernel2[3][3] = {
                    {-1, -2, -1},
                    {0, 0, 0},
                    {1, 2, 1}
    };
}
	__syncthreads();

    int initIteration = ((numberElements / totalThreads) * index) + offSet;
    int endIteration = initIteration + (numberElements / totalThreads) - 1;

    if (endIteration < (numberElements - offSet))
    {
        for (x = initIteration; x < endIteration; x = x + 3)
        {
            //Se debe realizar la operacion por cada uno de los colores RGB que se encuentran en cada pixel
            for (int f = 0; f < 3; f++)
            {
                float sum = 0.0;
                float sum2 = 0.0;
                float fsum = 0.0;
                //Se establece con estos dos fors la operacion de convolucion entre la matriz de la imagen y los kernels
                for (int k = -1; k <= 1; k++)
                {
                    for (int j = -1; j <= 1; j++)
                    {
                        sum = sum + Kernel[j + 1][k + 1] * d_imgGray[x + YoffSet * j + k*3 + f];
                        sum2 = sum2 + Kernel2[j + 1][k + 1] * d_imgGray[x + YoffSet * j + k*3 + f];
                    }
                }
                //Segun dicta el algoritmo se aplica la siguiente operacion
                fsum = ceilf(sqrt((sum * sum) + (sum2 * sum2)));
                //el valor resultante se substituye en el pixel correspondiente de la imagen objetivo
                d_imgSobel[x+f] = fsum;
            }
        }
    }

    __syncthreads();
}

__global__ void gray(unsigned char *d_imgOrig, unsigned char *d_imgGray, int rows, int numberElements, int totalThreads)
{

    int x;
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    int initIteration = (numberElements / totalThreads) * index;
    int endIteration = initIteration + (numberElements / totalThreads) - 1;

    if (endIteration < numberElements)
    {
        for (x = initIteration; x < endIteration; x = x + 3)
        {
            unsigned char r = d_imgOrig[x + 0];
            unsigned char g = d_imgOrig[x + 1];
            unsigned char b = d_imgOrig[x + 2];

            d_imgGray[x + 0] = r * 0.299f + g * 0.587f + b * 0.114f;
            d_imgGray[x + 1] = r * 0.299f + g * 0.587f + b * 0.114f;
            d_imgGray[x + 2] = r * 0.299f + g * 0.587f + b * 0.114f;
        }
    }
    __syncthreads();
}

int main(int argc, char *argv[])
{
    //-----------------------------------Variables------------------------------------//
    //errores de cuda
    hipError_t err = hipSuccess;
    int blocksPerGrid, threadsPerBlock;
    blocksPerGrid = atoi(argv[3]);
    threadsPerBlock = atoi(argv[4]);
    int totalThreads = blocksPerGrid * threadsPerBlock;
    //Definimos el conjunto de variables que utilizaremos para manejar las imagenes
    //Esto gracias al tipo de dato Mat que permite manejar la imagen como un objeto con atributos
    Mat imgOrig, imgSobel;
    unsigned char *h_imgOrig, *h_imgSobel, *h_imgGray;
    unsigned char *d_imgOrig, *d_imgSobel, *d_imgGray;
    int rows; 
    int cols; 
    //--------------------------------------------------------------------------------//

    //-----------------------------------Lectura imagen------------------------------------//
    //Se carga la imagen original como una imagen a color
    imgOrig = imread(argv[1], IMREAD_COLOR);
	
    //Se verifica que se cargo correctamente
    if (!imgOrig.data)
    {
        return -1;
    }

    //--------------------------------------------------------------------------------//

    //-----------------------------------Malloc------------------------------------//
    rows = imgOrig.rows;
    cols = imgOrig.cols;

    h_imgOrig = (unsigned char *)malloc(rows * cols * sizeof(unsigned char) * 3);
    unsigned char *rgb_image = imgOrig.data;

    //llenar el array de datos rgb del host
    int x = 0;
    for (x = 0; x < rows * cols * 3; x++)
        h_imgOrig[x] = rgb_image[x];

    size_t numElements = imgOrig.rows * imgOrig.cols;

    h_imgSobel = (unsigned char *)malloc(rows * cols * sizeof(unsigned char *)*3);
    h_imgGray = (unsigned char *)malloc(rows * cols * sizeof(unsigned char *)*3);
    //--------------------------------------------------------------------------------//

    //-----------------------------------CudaMalloc------------------------------------//

    err = hipMalloc(&d_imgOrig, sizeof(unsigned char) * numElements * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector imgOrig (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_imgSobel, sizeof(unsigned char) * numElements * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector vector imgSobel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset(d_imgSobel, 0, sizeof(unsigned char) * numElements * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set memory device vector imgSobel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc(&d_imgGray, sizeof(unsigned char) * numElements * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector imgGray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset(d_imgGray, 0, sizeof(unsigned char) * numElements * 3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set memory device vector imgGray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //--------------------------------------------------------------------------------//

    //-----------------------------------Tiempo------------------------------------//
    //Establecemos las variables de tiempo para las mediciones respectivas
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
    //--------------------------------------------------------------------------------//

    //-----------------------------------CudaMemcpy------------------------------------//
    err = hipMemcpy(d_imgOrig, h_imgOrig, sizeof(unsigned char) * numElements * 3, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector imgOrig from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //--------------------------------------------------------------------------------//

    //-----------------------------------__Global__------------------------------------//
    //Se hace llamado al metodo encargado de pasar la imagen original a escala de grises
    //como paso fundamental antes de aplicar sobel
    gray<<<blocksPerGrid, threadsPerBlock>>>(d_imgOrig, d_imgGray, rows, numElements * 3, totalThreads);
    //--------------------------------------------------------------------------------//

    //-----------------------------------CudaMemcpy - Results------------------------------------//
    err = hipMemcpy(h_imgGray, d_imgGray, sizeof(unsigned char) * numElements * 3, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector imgGray from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //--------------------------------------------------------------------------------//

    //-----------------------------------WriteGreyImg------------------------------------//
    //nombre de la imagen en escala de grises
    string string1((argv[1]));
    string1 = string1.substr(0, string1.size() - 4);
    string1 += "grayscale.png";
	
    //escribir imagen en escala de grises
    cv::Mat greyData(rows, cols, CV_8UC3, (void *)h_imgGray);
    cv::imwrite(string1, greyData);
    //--------------------------------------------------------------------------------//

    //-----------------------------------__Global__------------------------------------//

    //Se llama a la funcion que realiza el procedimiento para hallar sobel
    sobel<<<blocksPerGrid, threadsPerBlock>>>(d_imgGray, d_imgSobel,cols, rows, numElements * 3, totalThreads);
    //--------------------------------------------------------------------------------//

    //-----------------------------------CudaMemcpy - Results------------------------------------//
    err = hipMemcpy(h_imgSobel, d_imgSobel, sizeof(unsigned char) * numElements * 3, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_imgSobel from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //--------------------------------------------------------------------------------//

    //Se guarda la imagen correspondiente a sobel
    cv::Mat sobelData(rows, cols, CV_8UC3, (void *)h_imgSobel);
    cv::imwrite(argv[2], sobelData);

    //-----------------------------------CudaFree------------------------------------//
    err = hipFree(d_imgOrig);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_imgOrig (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_imgGray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_imgGray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_imgSobel);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device d_imgSobel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    //--------------------------------------------------------------------------------//

    //-----------------------------------Tiempo - Final------------------------------------//
    //Se finaliza el registro del tiempo
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);

    //escritura de los tiempos en el txt
    ofstream myfile;
    myfile.open("tiempos.txt", std::ios_base::app);
    myfile << "Imagen: " << argv[1] << " - ";
    myfile << "Tiempo: " << tval_result.tv_sec << "." << tval_result.tv_usec << " s - ";
    myfile << "Bloques: " << blocksPerGrid << " - Hilos por bloque: " << threadsPerBlock << "\n";
    myfile.close();
    //---------------------------------------------------------------------------------//

    return 0;
}
